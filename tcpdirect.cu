
#include <hip/hip_runtime.h>
#ifdef WITH_TCPDIRECT
#include <cuda.h>
#include <cuda_runtime.h>

#include <asm-generic/errno-base.h>
#include <asm-generic/socket.h>
#include <errno.h>
#include <fcntl.h>
#include <linux/if.h>
#include <stdlib.h>
#include <sys/ioctl.h>
#include <sys/mman.h>
#include <sys/socket.h>
#include <unistd.h>
#include <linux/dma-buf.h>

#include <memory>
#include <cstdint>
#include <cstdio>
#include <string>
#include <vector>

#include "tcpdirect.h"
#include "logging.h"
#include "flow.h"
#include "thread.h"

#ifndef MSG_ZEROCOPY
#define MSG_ZEROCOPY	0x4000000
#endif

#define LAST_PRIME 111

#define MIN_RX_BUFFER_TOTAL_SIZE (1 << 28)
#define GPUMEM_ALIGNMENT (1UL << 21)
#define GPUMEM_MINSZ 0x400000
#define PAGE_SHIFT (12)
#define PAGE_SIZE (1 << PAGE_SHIFT)

#define multiplier (1 << 16)

#define TEST_PREFIX "ncdevmem"
#define NUM_PAGES 16000

/* missing definitions in mman-linux.h */
#ifndef MFD_ALLOW_SEALING
#define MFD_ALLOW_SEALING 2U
#endif

/* GRTE libraries from google3 already define the following */
#ifndef F_SEAL_SHRINK
#define F_SEAL_SHRINK 2U
#endif
#ifndef F_ADD_SEALS
#define F_ADD_SEALS 1033U
#endif
#ifndef F_GET_SEALS
#define F_GET_SEALS 1034U
#endif

#define MSG_SOCK_DEVMEM 0x2000000
#define SO_DEVMEM_DONTNEED 97
#define SO_DEVMEM_HEADER 98
#define SCM_DEVMEM_HEADER SO_DEVMEM_HEADER
#define SO_DEVMEM_OFFSET 99
#define SCM_DEVMEM_OFFSET SO_DEVMEM_OFFSET

struct dma_buf_create_pages_info {
  __u64 pci_bdf[3];
  __s32 dma_buf_fd;
  __s32 create_page_pool;
};

struct dma_buf_pages_bind_rx_queue {
  char ifname[IFNAMSIZ];
  __u32 rxq_idx;
};

#define DMA_BUF_CREATE_PAGES \
  _IOW(DMA_BUF_BASE, 2, struct dma_buf_create_pages_info)

#define DMA_BUF_PAGES_BIND_RX \
  _IOW(DMA_BUF_BASE, 3, struct dma_buf_pages_bind_rx_queue)

// devmemvec represents a fragment of payload that is received on the socket.
struct devmemvec {
  // frag_offset is the offset in the registered memory.
  __u32 frag_offset;
  // frag size is the size of the payload.
  __u32 frag_size;
  // frag_token is an identifier for this fragment and it can be used to return
  // the memory back to kernel.
  __u32 frag_token;
};

// devmemtoken represents a range of tokens. It is used to return the fragment
// memory back to the kernel.
struct devmemtoken {
  __u32 token_start;
  __u32 token_count;
};

struct udmabuf_create {
  uint32_t memfd;
  uint32_t flags;
  uint64_t offset;
  uint64_t size;
};
#define UDMABUF_CREATE _IOW('u', 0x42, struct udmabuf_create)

int memfd_create(const char *name, unsigned int flags)
{
	return syscall(__NR_memfd_create, name, flags);
}

/* Fills buf of size n with a repeating sequence of 1 to 111 inclusive
 */
void fill_tx_buffer(void *buf, size_t n) {
#define BUFSIZE 3996
  unsigned char src_buf[BUFSIZE];
  int ptr = 0, i = 0;

  while (i < BUFSIZE) {
    src_buf[i] = (i % LAST_PRIME) + 1;
    i++;
  }

  while (ptr*BUFSIZE + BUFSIZE < n) {
    cudaMemcpy((char *)buf + ptr*BUFSIZE, &src_buf, BUFSIZE, cudaMemcpyHostToDevice);
    ptr++;
  }

  i = ptr*BUFSIZE;
  while (i < n) {
    cudaMemset((char *)buf + i, (i % LAST_PRIME) + 1, 1);
    i++;
  }
}

int tcpdirect_setup_socket(int socket) {
  const int one = 1;
  if (setsockopt(socket, SOL_SOCKET, SO_REUSEADDR, &one, sizeof(one))
      || setsockopt(socket, SOL_SOCKET, SO_REUSEPORT, &one, sizeof(one))
      || setsockopt(socket, SOL_SOCKET, SO_ZEROCOPY, &one, sizeof(one))
     ) {
    perror("tcpdirect_setup_socket");
    exit(EXIT_FAILURE);
  }

  return 0;
}

int get_gpumem_dmabuf_pages_fd(const std::string& gpu_pci_addr,
                               const std::string& nic_pci_addr, void* gpu_mem,
                               size_t gpu_mem_sz, int* dma_buf_fd, bool is_client) {
  int err, ret;

  cuMemGetHandleForAddressRange((void*)dma_buf_fd, (CUdeviceptr)gpu_mem,
                                gpu_mem_sz, CU_MEM_RANGE_HANDLE_TYPE_DMA_BUF_FD,
                                0);

  if (*dma_buf_fd < 0) {
    perror("cuMemGetHandleForAddressRange() failed!: ");
    exit(EXIT_FAILURE);
  }

  printf("Registered dmabuf region 0x%p of %lu Bytes\n",
      gpu_mem, gpu_mem_sz);

  struct dma_buf_create_pages_info frags_create_info;
  frags_create_info.dma_buf_fd = *dma_buf_fd;
  frags_create_info.create_page_pool = is_client ? 0 : 1;

  uint16_t pci_bdf[3];
  ret = sscanf(nic_pci_addr.c_str(), "0000:%hx:%hx.%hx", &pci_bdf[0],
               &pci_bdf[1], &pci_bdf[2]);
  frags_create_info.pci_bdf[0] = pci_bdf[0];
  frags_create_info.pci_bdf[1] = pci_bdf[1];
  frags_create_info.pci_bdf[2] = pci_bdf[2];
  if (ret != 3) {
    err = -EINVAL;
    goto err_close_dmabuf;
  }

  ret = ioctl(*dma_buf_fd, DMA_BUF_CREATE_PAGES, &frags_create_info);
  if (ret < 0) {
    perror("Error getting dma_buf frags: ");
    err = -EIO;
    goto err_close_dmabuf;
  }
  return ret;

err_close_dmabuf:
  close(*dma_buf_fd);
  return err;
}

int tcpdirect_cuda_setup_alloc(const struct options *opts, void **f_mbuf, struct thread *t)
{
  bool is_client = opts->client;
  int ret;
  void *gpu_tx_mem_;
  int gpu_mem_fd_;
  int dma_buf_fd_;
  int q_start = opts->queue_start;
  int q_num = opts->queue_num;
  struct tcpdirect_cuda_mbuf *tmbuf;
  const char *gpu_pci_addr = opts->tcpd_gpu_pci_addr;  // "0000:04:00.0"
  const char *nic_pci_addr = opts->tcpd_nic_pci_addr;  // "0000:06:00.0"
  size_t alloc_size = opts->tcpdirect_phys_len;

  tmbuf =
    (struct tcpdirect_cuda_mbuf *)calloc(1, sizeof(struct tcpdirect_cuda_mbuf));
  if (!tmbuf) {
    exit(EXIT_FAILURE);
  }

  if (alloc_size % GPUMEM_ALIGNMENT != 0) {
    alloc_size += GPUMEM_ALIGNMENT - (alloc_size % GPUMEM_ALIGNMENT);
  }

  // unnecessary if CUDA_VISIBLE_DEVICES env var is set
  // ret = cudaSetDevice(opts->tcpdirect_gpu_idx);
  // if (ret != 0) {
  //   printf("cudaSetDevice failed: index %i", opts->tcpdirect_gpu_idx);
  //   exit(70);
  // }

  cudaMalloc(&gpu_tx_mem_, alloc_size);
  if (is_client && opts->tcpd_validate) {
          fill_tx_buffer(gpu_tx_mem_, alloc_size);
          cudaDeviceSynchronize();
  }
  unsigned int flag = 1;
  cuPointerSetAttribute(&flag,
                        CU_POINTER_ATTRIBUTE_SYNC_MEMOPS,
                        (CUdeviceptr)gpu_tx_mem_);

  gpu_mem_fd_ = get_gpumem_dmabuf_pages_fd(gpu_pci_addr, nic_pci_addr,
                                           gpu_tx_mem_, alloc_size,
                                           &dma_buf_fd_, is_client);

  if (gpu_mem_fd_ < 0) {
    printf("get_gpumem_dmabuf_pages_fd() failed!: ");
    exit(71);
  }

  if (!is_client) {
    int num_queues = q_start + (t->index % q_num);
    printf("Bind to queue %i\n", num_queues);
    struct dma_buf_pages_bind_rx_queue bind_cmd;

    strcpy(bind_cmd.ifname, opts->tcpdirect_link_name);
    bind_cmd.rxq_idx = num_queues;

    ret = ioctl(gpu_mem_fd_, DMA_BUF_PAGES_BIND_RX, &bind_cmd);
    if (ret < 0) {
      printf("%s: [FAIL, bind fail queue=%d]\n", TEST_PREFIX,
            num_queues);
      exit(78);
    }

    // copied from socket.c#socket_connect_one()
    int flow_idx = (t->flow_first + t->flow_count);
    int src_port = flow_idx + opts->source_port;
    int dst_port = flow_idx + atoi(opts->port);

    char flow_steer_cmd[512];
    sprintf(flow_steer_cmd,
            "ethtool -N %s flow-type tcp4 src-ip %s dst-ip %s src-port %i dst-port %i queue %i",
            opts->tcpdirect_link_name, opts->tcpdirect_src_ip, opts->tcpdirect_dst_ip, src_port, dst_port, num_queues);
    ret = system(flow_steer_cmd);

    // only running the below ethtool commands after last thread/flow is setup
    if (flow_idx + t->flow_limit >= opts->num_flows) {
      char ethtool_cmd[512];
      sprintf(ethtool_cmd, "ethtool --set-priv-flags %s enable-strict-header-split on", opts->tcpdirect_link_name);
      ret = ret | system(ethtool_cmd);
      sprintf(ethtool_cmd, "ethtool --set-priv-flags %s enable-header-split on", opts->tcpdirect_link_name);
      ret = ret | system(ethtool_cmd);
      sprintf(ethtool_cmd, "ethtool --set-rxfh-indir %s equal 8", opts->tcpdirect_link_name);
      ret = ret | system(ethtool_cmd);
      printf("ethtool cmds returned %i, sleeping 1...\n", ret);
      sleep(1);
    }
  }

  *f_mbuf = tmbuf;
  tmbuf->gpu_mem_fd_ = gpu_mem_fd_;
  tmbuf->dma_buf_fd_ = dma_buf_fd_;
  tmbuf->gpu_tx_mem_ = gpu_tx_mem_;
  tmbuf->cpy_buffer = malloc(opts->buffer_size);
  tmbuf->vectors = new std::vector<devmemvec>();
  tmbuf->tokens = new std::vector<devmemtoken>();
  tmbuf->bytes_received = 0;
  tmbuf->bytes_sent = 0;
  return 0;
}

int udmabuf_setup_alloc(const struct options *opts, void **f_mbuf) {
  bool is_client = opts->client;
  int devfd;
  int memfd;
  int buf;
  int buf_pages;
  int ret;
  size_t size = opts->tcpdirect_phys_len;

  struct tcpdirect_udma_mbuf *tmbuf;
  struct dma_buf_create_pages_info pages_create_info;
  struct udmabuf_create create;

  if (f_mbuf == NULL) return ENOMEM;

  if (*f_mbuf) return 0;

  tmbuf = (struct tcpdirect_udma_mbuf *)calloc(1, sizeof(struct tcpdirect_udma_mbuf));
  if (!tmbuf) {
    exit(EXIT_FAILURE);
  }

  devfd = open("/dev/udmabuf", O_RDWR);
  if (devfd < 0) {
    printf("%s: [skip,no-udmabuf: Unable to access DMA buffer device file]\n",
           TEST_PREFIX);
    exit(70);
  }

  memfd = memfd_create("udmabuf-test", MFD_ALLOW_SEALING);
  if (memfd < 0) {
    printf("%s: [skip,no-memfd]\n", TEST_PREFIX);
    exit(72);
  }

  ret = fcntl(memfd, F_ADD_SEALS, F_SEAL_SHRINK);
  if (ret < 0) {
    printf("%s: [skip,fcntl-add-seals]\n", TEST_PREFIX);
    exit(73);
  }

  ret = ftruncate(memfd, size);
  if (ret == -1) {
    printf("%s: [FAIL,memfd-truncate]\n", TEST_PREFIX);
    exit(74);
  }

  memset(&create, 0, sizeof(create));

  create.memfd = memfd;
  create.offset = 0;
  create.size = size;
  printf("size=%lu\n", size);
  buf = ioctl(devfd, UDMABUF_CREATE, &create);
  if (buf < 0) {
    printf("%s: [FAIL, create udmabuf]\n", TEST_PREFIX);
    exit(75);
  }

  pages_create_info.dma_buf_fd = buf;
  pages_create_info.create_page_pool = is_client ? 0 : 1;

  /* TODO: hardcoded NIC pci address */
  // "0000:06:00.0"
  ret = sscanf(opts->tcpd_nic_pci_addr, "0000:%llx:%llx.%llx",
         &pages_create_info.pci_bdf[0],
         &pages_create_info.pci_bdf[1],
         &pages_create_info.pci_bdf[2]);

  if (ret != 3) {
    printf("%s: [FAIL, parse fail]\n", TEST_PREFIX);
    exit(76);
  }

  buf_pages = ioctl(buf, DMA_BUF_CREATE_PAGES, &pages_create_info);
  if (buf_pages < 0) {
    perror("ioctl DMA_BUF_CREATE_PAGES: [FAIL, create pages fail]\n");
    exit(77);
  }

  if (!is_client) {
    /* TODO hardcoded num_queues */
    int num_queues = 15;
    struct dma_buf_pages_bind_rx_queue bind_cmd;

    strcpy(bind_cmd.ifname, "eth1");
    bind_cmd.rxq_idx = num_queues;

    ret = ioctl(buf_pages, DMA_BUF_PAGES_BIND_RX, &bind_cmd);
    if (ret < 0) {
      printf("%s: [FAIL, bind fail queue=%d]\n", TEST_PREFIX,
            num_queues);
      exit(78);
    }

    system("ethtool --set-priv-flags eth1 enable-header-split on");
    system("ethtool --set-priv-flags eth1 enable-header-split off");
	  system("ethtool --set-priv-flags eth1 enable-header-split on");
    sleep(1);
    printf("toggled header-split\n");
  }

  struct dma_buf_sync sync = { 0 };
  sync.flags = DMA_BUF_SYNC_WRITE | DMA_BUF_SYNC_START;
  ioctl(buf, DMA_BUF_IOCTL_SYNC, &sync);

  *f_mbuf = tmbuf;

  tmbuf->devfd = devfd;
  tmbuf->memfd = memfd;
  tmbuf->buf = buf;
  tmbuf->buf_pages = buf_pages;
  return 0;
}

int tcpdirect_udma_send(int socket, void *f_mbuf, size_t n, int flags) {
  int buf_pages, buf;
  struct iovec iov;
  struct msghdr *msg;
  struct cmsghdr *cmsg;
  char buf_dummy[n];
  char offsetbuf[CMSG_SPACE(sizeof(uint32_t) * 2)];
  struct tcpdirect_udma_mbuf *tmbuf;

  if (!f_mbuf) return -1;

  tmbuf = (struct tcpdirect_udma_mbuf *)f_mbuf;
  buf_pages = tmbuf->buf_pages;
  buf = tmbuf->buf;
  msg = &tmbuf->msg;

  struct dma_buf_sync sync = { 0 };
  sync.flags = DMA_BUF_SYNC_WRITE | DMA_BUF_SYNC_START;
  ioctl(buf, DMA_BUF_IOCTL_SYNC, &sync);

  char *buf_mem = NULL;
  buf_mem = (char *)mmap(NULL, n, PROT_READ | PROT_WRITE, MAP_SHARED, buf, 0);
  if (buf_mem == MAP_FAILED) {
    perror("mmap()");
    exit(1);
  }

  memcpy(buf_mem, buf_dummy, n);

  sync.flags = DMA_BUF_SYNC_WRITE | DMA_BUF_SYNC_END;
  ioctl(buf, DMA_BUF_IOCTL_SYNC, &sync);

  munmap(buf_mem, n);

  memset(msg, 0, sizeof(struct msghdr));
  // memset(cmsg, 0, sizeof(struct cmsghdr));

  iov.iov_base = buf_dummy;
  iov.iov_len = n;

  msg->msg_iov = &iov;
  msg->msg_iovlen = 1;

  msg->msg_control = offsetbuf;
  msg->msg_controllen = sizeof(offsetbuf);

  cmsg = CMSG_FIRSTHDR(msg);
  cmsg->cmsg_level = SOL_SOCKET;
  cmsg->cmsg_type = SCM_DEVMEM_OFFSET;
  cmsg->cmsg_len = CMSG_LEN(sizeof(int) * 2);
  *((int*)CMSG_DATA(cmsg)) = buf_pages;
  ((int*)CMSG_DATA(cmsg))[1] = 0;

  ssize_t bytes_sent = sendmsg(socket, msg, MSG_ZEROCOPY);
  if (bytes_sent < 0 && errno != EWOULDBLOCK && errno != EAGAIN) {
    perror("sendmsg() error: ");
    exit(EXIT_FAILURE);
  }

  if (bytes_sent == 0) {
    perror("sendmsg() sent 0 bytes. Something is wrong.\n");
    exit(EXIT_FAILURE);
  }

  return bytes_sent;
}

int tcpdirect_send(int socket, void *buf, size_t n, int flags) {
  int gpu_mem_fd_;
  struct iovec iov;
  struct msghdr msg;
  struct cmsghdr *cmsg;
  char offsetbuf[CMSG_SPACE(sizeof(uint32_t) * 2)];
  struct tcpdirect_cuda_mbuf *tmbuf;

  if (!buf) return -1;

  tmbuf = (struct tcpdirect_cuda_mbuf *)buf;
  gpu_mem_fd_ = tmbuf->gpu_mem_fd_;

  memset(&msg, 0, sizeof(msg));
  // memset(cmsg, 0, sizeof(struct cmsghdr));

  iov.iov_base = NULL;
  iov.iov_len = n - tmbuf->bytes_sent;

  msg.msg_iov = &iov;
  msg.msg_iovlen = 1;

  msg.msg_control = offsetbuf;
  msg.msg_controllen = sizeof(offsetbuf);

  cmsg = CMSG_FIRSTHDR(&msg);
  cmsg->cmsg_level = SOL_SOCKET;
  cmsg->cmsg_type = SCM_DEVMEM_OFFSET;
  cmsg->cmsg_len = CMSG_LEN(sizeof(int) * 2);
  *((int*)CMSG_DATA(cmsg)) = gpu_mem_fd_;
  ((int *)CMSG_DATA(cmsg))[1] = (int)tmbuf->bytes_sent;

  ssize_t bytes_sent = sendmsg(socket, &msg, MSG_ZEROCOPY | MSG_DONTWAIT);
  if (bytes_sent < 0 && errno != EWOULDBLOCK && errno != EAGAIN) {
    perror("sendmsg() error: ");
    exit(EXIT_FAILURE);
  }

  if (bytes_sent == 0) {
    perror("sendmsg() sent 0 bytes. Something is wrong.\n");
    exit(EXIT_FAILURE);
  }

  tmbuf->bytes_sent += bytes_sent;
  if (tmbuf->bytes_sent == n)
    tmbuf->bytes_sent = 0;

  return bytes_sent;
}

int tcpdirect_recv(int socket, void *f_mbuf, size_t n, int flags, struct thread *t) {
  struct iovec iov;
  struct msghdr msg_local;
  struct msghdr *msg;
  struct tcpdirect_cuda_mbuf *tmbuf;
  int ret, client_fd; // buf
  int buffer_size = n;
  size_t total_received = 0;
  unsigned char *cpy_buffer;
  const struct options *opts = t->opts;
  std::vector<devmemvec> *vectors;
  std::vector<devmemtoken> *tokens;

  if (!f_mbuf) return -1;

  tmbuf = (struct tcpdirect_cuda_mbuf *)f_mbuf;
  cpy_buffer = (unsigned char *)tmbuf->cpy_buffer;
  vectors = (std::vector<devmemvec> *)tmbuf->vectors;
  tokens = (std::vector<devmemtoken> *)tmbuf->tokens;

  client_fd = socket;

  char buf_dummy[n];
  // char offsetbuf[CMSG_SPACE(sizeof(uint32_t) * 128)];
  char offsetbuf[CMSG_SPACE(sizeof(int) * 1000)];
  msg = &msg_local;

  memset(msg, 0, sizeof(struct msghdr));

  iov.iov_base = buf_dummy;
  iov.iov_len = n - tmbuf->bytes_received;
  msg->msg_iov = &iov;
  msg->msg_iovlen = 1;

  msg->msg_control = offsetbuf;
  msg->msg_controllen = sizeof(offsetbuf);

  // char *buf_mem = NULL;

  if (msg->msg_flags & MSG_CTRUNC) {
    printf("fatal, cmsg truncated, current msg_controllen\n");
 }

  ssize_t received = recvmsg(socket, msg, MSG_SOCK_DEVMEM | MSG_DONTWAIT);
  if (received < 0 && (errno == EAGAIN || errno == EWOULDBLOCK)) {
  } else if (received < 0) {
    printf("%s %d\n", __func__, __LINE__);
    return -1;
  } else if (received == 0) {
    printf("Client exited\n");
  }

  struct cmsghdr *cm = NULL;
  struct devmemvec *devmemvec = NULL;
  for (cm = CMSG_FIRSTHDR(msg); cm; cm = CMSG_NXTHDR(msg, cm)) {
    if (cm->cmsg_level != SOL_SOCKET ||
        (cm->cmsg_type != SCM_DEVMEM_OFFSET &&
          cm->cmsg_type != SCM_DEVMEM_HEADER)) {
      continue;
    }

    devmemvec = (struct devmemvec *)CMSG_DATA(cm);

    if (cm->cmsg_type == SCM_DEVMEM_HEADER) {
      // TODO: process data copied from skb's linear
      // buffer.
      fprintf(stderr, "\n\nSCM_DEVMEM_HEADER. devmemvec->frag_size=%u\n",
              devmemvec->frag_size);
      exit(1);
    }

    struct devmemtoken token = { devmemvec->frag_token, 1 };

    // struct dma_buf_sync sync = { 0 };
    // sync.flags = DMA_BUF_SYNC_READ | DMA_BUF_SYNC_START;
    // ioctl(buf, DMA_BUF_IOCTL_SYNC, &sync);

    // buf_mem = (char *)mmap(NULL, n, PROT_READ | PROT_WRITE,
    //                MAP_SHARED, buf, 0);
    // if (buf_mem == MAP_FAILED) {
    //   perror("mmap()");
    //   exit(1);
    // }
    total_received += devmemvec->frag_size;
    // printf("\n\nreceived frag_page=%u, in_page_offset=%u,"
    //         " frag_offset=%u, frag_size=%u, token=%u"
    //         " total_received=%lu\n",
    //         devmemvec->frag_offset >> PAGE_SHIFT,
    //         devmemvec->frag_offset % PAGE_SIZE,
    //         devmemvec->frag_offset, devmemvec->frag_size,
    //         devmemvec->frag_token,
    //         total_received);

    // sync.flags = DMA_BUF_SYNC_READ | DMA_BUF_SYNC_END;
    // ioctl(buf, DMA_BUF_IOCTL_SYNC, &sync);
    vectors->emplace_back(*devmemvec);
    tokens->push_back(token);
    // munmap(buf_mem, n);
  }

  tmbuf->bytes_received += received;

  /* Once we've received fragments totaling buffer_size, we can copy from the
   * CUDA buffer to a user-space buffer, and free the fragments in the CUDA
   * buffer.
  */
  if (tmbuf->bytes_received == buffer_size) {
    /* There is a performance impact when we cudaMemcpy from the CUDA buffer to
     * the userspace buffer, so it's gated by a flag
     */
    if (opts->tcpd_rx_cpy || opts->tcpd_validate) {
      for (int idx = 0; idx < vectors->size(); idx++) {
        struct devmemvec vec = (*vectors)[idx];
        struct devmemtoken token = (*tokens)[idx];

        /* copy each fragment to the cpy_buffer in order, i.e.
         * 1st fragment will occuply bytes [0-4095], 2nd fragment will
         * occupy bytes [4096-8191], etc.
         */
        cudaMemcpy(cpy_buffer + (vec.frag_token - 1) * PAGE_SIZE,
                   (char *)tmbuf->gpu_tx_mem_ + vec.frag_offset,
                   vec.frag_size,
                   cudaMemcpyDeviceToHost);
      }

      /* Ensure the sequence is what we expect:
       * a repeating sequence of 1 to LAST_PRIME inclusive
       */
      if (opts->tcpd_validate) {
        cudaDeviceSynchronize();
        int i = 0;
        int expected_val;
        while (i < buffer_size) {
          expected_val = (i % LAST_PRIME) + 1;
          if (cpy_buffer[i] != expected_val) {
            printf("Thread %i - incorrect byte %i, expected %i, got %i\n",
                  t->index,
                  i,
                  expected_val,
                  cpy_buffer[i]);
            break;
          }
          i++;
        }
      }
    }

    ret = setsockopt(client_fd, SOL_SOCKET,
                      SO_DEVMEM_DONTNEED, tokens->data(),
                      tokens->size() * sizeof(devmemtoken));
    if (ret) {
      perror("DONTNEED failed");
      exit(1);
    }
    vectors->clear();
    tokens->clear();
    tmbuf->bytes_received = 0;
  }
  return total_received;
}

int cuda_flow_cleanup(void *f_mbuf) {
  struct tcpdirect_cuda_mbuf *t_mbuf = (struct tcpdirect_cuda_mbuf *)f_mbuf;
  close(t_mbuf->gpu_mem_fd_);
  close(t_mbuf->dma_buf_fd_);
  cudaFree(t_mbuf->gpu_tx_mem_);
  free(t_mbuf->cpy_buffer);
  free(t_mbuf->tokens);
  free(t_mbuf->vectors);
  return 0;
}
#endif /* #ifdef WITH_TCPDIRECT */
